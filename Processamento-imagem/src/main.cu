#include "hip/hip_runtime.h"
#include "imagem_process.cu"
#include <time.h>

char path_abs[100] = "/home/grupo04/"; //informe path base de localização dos diretorios


int main(int argc, char const *argv[])
{

	clock_t tempo;
    tempo = clock();
	char path_memorial_input[150], path_memorial_out_gray[150], path_memorial_out_toggle[150], path_memorial_out_inverte[150];
	char path_vista_diagonal_input[150], path_vista_diagonal_out_gray[150], path_vista_diagonal_out_toggle[150], path_vista_diagonal_out_inverte[150];
	char path_vista_frontal_input[150], path_vista_frontal_out_gray[150], path_vista_frontal_out_toggle[150], path_vista_frontal_out_inverte[150];
	char path_vista_fundo_input[150], path_vista_fundo_out_gray[150], path_vista_fundo_out_toggle[150], path_vista_fundo_out_inverte[150];
	char path_vista_lateral_direita_input[150], path_vista_lateral_direita_out_gray[150], path_vista_lateral_direita_out_toggle[150], path_vista_lateral_direita_out_inverte[150];
	char path_vista_lateral_esquerda_input[150], path_vista_lateral_esquerda_out_gray[150], path_vista_lateral_esquerda_out_toggle[150], path_vista_lateral_esquerda_out_inverte[150];
	char path_vista_superior_input[150], path_vista_superior_out_gray[150], path_vista_superior_out_toggle[150], path_vista_superior_out_inverte[150];

	/* Path's*/	
	strcpy(path_memorial_input, path_abs);
	strcat(path_memorial_input, "Processamento-imagem/imgs/memorial_img/memorial.ppm");
	strcpy(path_memorial_out_gray, path_abs);
	strcat(path_memorial_out_gray, "Processamento-imagem/imgs/memorial_img/memorial_out_gray.ppm");
	strcpy(path_memorial_out_toggle, path_abs);
	strcat(path_memorial_out_toggle, "Processamento-imagem/imgs/memorial_img/memorial_out_toggle.ppm");
	strcpy(path_memorial_out_inverte, path_abs);
	strcat(path_memorial_out_inverte, "Processamento-imagem/imgs/memorial_img/memorial_out_inverte.ppm");

	strcpy(path_vista_diagonal_input, path_abs);
	strcat(path_vista_diagonal_input, "Processamento-imagem/imgs/vista_diagonal_img/vista_diagonal.ppm");
	strcpy(path_vista_diagonal_out_gray, path_abs);
	strcat(path_vista_diagonal_out_gray, "Processamento-imagem/imgs/vista_diagonal_img/vista_diagonal_out_gray.ppm");
	strcpy(path_vista_diagonal_out_toggle, path_abs);
	strcat(path_vista_diagonal_out_toggle, "Processamento-imagem/imgs/vista_diagonal_img/vista_diagonal_out_toggle.ppm");
	strcpy(path_vista_diagonal_out_inverte, path_abs);
	strcat(path_vista_diagonal_out_inverte, "Processamento-imagem/imgs/vista_diagonal_img/vista_diagonal_out_inverte.ppm");

	strcpy(path_vista_frontal_input, path_abs);
	strcat(path_vista_frontal_input, "Processamento-imagem/imgs/vista_frontal_img/vista_frontal.ppm");
	strcpy(path_vista_frontal_out_gray, path_abs);
	strcat(path_vista_frontal_out_gray, "Processamento-imagem/imgs/vista_frontal_img/vista_frontal_out_gray.ppm");
	strcpy(path_vista_frontal_out_toggle, path_abs);
	strcat(path_vista_frontal_out_toggle, "Processamento-imagem/imgs/vista_frontal_img/vista_frontal_out_toggle.ppm");
	strcpy(path_vista_frontal_out_inverte, path_abs);
	strcat(path_vista_frontal_out_inverte, "Processamento-imagem/imgs/vista_frontal_img/vista_frontal_out_inverte.ppm");

	strcpy(path_vista_fundo_input, path_abs);
	strcat(path_vista_fundo_input, "Processamento-imagem/imgs/vista_fundo_img/vista_fundo.ppm");
	strcpy(path_vista_fundo_out_gray, path_abs);
	strcat(path_vista_fundo_out_gray, "Processamento-imagem/imgs/vista_fundo_img/vista_fundo_out_gray.ppm");
	strcpy(path_vista_fundo_out_toggle, path_abs);
	strcat(path_vista_fundo_out_toggle, "Processamento-imagem/imgs/vista_fundo_img/vista_fundo_out_toggle.ppm");
	strcpy(path_vista_fundo_out_inverte, path_abs);
	strcat(path_vista_fundo_out_inverte, "Processamento-imagem/imgs/vista_fundo_img/vista_fundo_out_inverte.ppm");

	strcpy(path_vista_lateral_direita_input, path_abs);
	strcat(path_vista_lateral_direita_input, "Processamento-imagem/imgs/vista_lateral_direita_img/vista_lateral_direita.ppm");
	strcpy(path_vista_lateral_direita_out_gray, path_abs);
	strcat(path_vista_lateral_direita_out_gray, "Processamento-imagem/imgs/vista_lateral_direita_img/vista_lateral_direita_out_gray.ppm");
	strcpy(path_vista_lateral_direita_out_toggle, path_abs);
	strcat(path_vista_lateral_direita_out_toggle, "Processamento-imagem/imgs/vista_lateral_direita_img/vista_lateral_direita_out_toggle.ppm");
	strcpy(path_vista_lateral_direita_out_inverte, path_abs);
	strcat(path_vista_lateral_direita_out_inverte, "Processamento-imagem/imgs/vista_lateral_direita_img/vista_lateral_direita_out_inverte.ppm");

	strcpy(path_vista_lateral_esquerda_input, path_abs);
	strcat(path_vista_lateral_esquerda_input, "Processamento-imagem/imgs/vista_lateral_esquerda_img/vista_lateral_esquerda.ppm");
	strcpy(path_vista_lateral_esquerda_out_gray, path_abs);
	strcat(path_vista_lateral_esquerda_out_gray, "Processamento-imagem/imgs/vista_lateral_esquerda_img/vista_lateral_esquerda_out_gray.ppm");
	strcpy(path_vista_lateral_esquerda_out_toggle, path_abs);
	strcat(path_vista_lateral_esquerda_out_toggle, "Processamento-imagem/imgs/vista_lateral_esquerda_img/vista_lateral_esquerda_out_toggle.ppm");
	strcpy(path_vista_lateral_esquerda_out_inverte, path_abs);
	strcat(path_vista_lateral_esquerda_out_inverte, "Processamento-imagem/imgs/vista_lateral_esquerda_img/vista_lateral_esquerda_out_inverte.ppm");

	strcpy(path_vista_superior_input, path_abs);
	strcat(path_vista_superior_input, "Processamento-imagem/imgs/vista_superior_img/vista_superior.ppm");
	strcpy(path_vista_superior_out_gray, path_abs);
	strcat(path_vista_superior_out_gray, "Processamento-imagem/imgs/vista_superior_img/vista_superior_out_gray.ppm");
	strcpy(path_vista_superior_out_toggle, path_abs);
	strcat(path_vista_superior_out_toggle, "Processamento-imagem/imgs/vista_superior_img/vista_superior_out_toggle.ppm");
	strcpy(path_vista_superior_out_inverte, path_abs);
	strcat(path_vista_superior_out_inverte, "Processamento-imagem/imgs/vista_superior_img/vista_superior_out_inverte.ppm");

	/*Datas*/
	unsigned long len_fptr;
	unsigned long len_fptr_vista_diagonal;
	unsigned long len_fptr_vista_frontal;
	unsigned long len_fptr_vista_fundo;
	unsigned long len_fptr_vista_lateral_direita;
	unsigned long len_fptr_vista_lateral_esquerda;
	unsigned long len_fptr_vista_superior;

	unsigned char * datas = read_image_input(path_memorial_input, &len_fptr);
	unsigned char * datas_vista_diagonal = read_image_input(path_vista_diagonal_input, &len_fptr_vista_diagonal);
	unsigned char * datas_vista_frontal = read_image_input(path_vista_frontal_input, &len_fptr_vista_frontal);
	unsigned char * datas_vista_fundo = read_image_input(path_vista_fundo_input, &len_fptr_vista_fundo);
	unsigned char * datas_vista_lateral_direita = read_image_input(path_vista_lateral_direita_input, &len_fptr_vista_lateral_direita);
	unsigned char * datas_vista_lateral_esquerda = read_image_input(path_vista_lateral_esquerda_input, &len_fptr_vista_lateral_esquerda);
	unsigned char * datas_vista_superior = read_image_input(path_vista_superior_input, &len_fptr_vista_superior);

	/*Efeitos*/
	unsigned char * data_gray = tonalidade_gray(datas, len_fptr, 7);
	unsigned char * data_gray_toggle = tonalidade_toggle_gray(datas, len_fptr, 512, 7);
	unsigned char * data_inverte = inverte_image(datas, len_fptr, 7);

	unsigned char * data_gray_vista_diagonal = tonalidade_gray(datas_vista_diagonal, len_fptr_vista_diagonal, 9);
	unsigned char * data_gray_toggle_vista_diagonal = tonalidade_toggle_gray(datas_vista_diagonal, len_fptr_vista_diagonal, 3840, 9);
	unsigned char * data_inverte_vista_diagonal = inverte_image(datas_vista_diagonal, len_fptr_vista_diagonal, 9);

	unsigned char * data_gray_vista_frontal = tonalidade_gray(datas_vista_frontal, len_fptr_vista_frontal, 9);
	unsigned char * data_gray_toggle_vista_frontal = tonalidade_toggle_gray(datas_vista_frontal, len_fptr_vista_frontal, 3840, 9);
	unsigned char * data_inverte_vista_frontal = inverte_image(datas_vista_frontal, len_fptr_vista_frontal, 9);

	unsigned char * data_gray_vista_fundo = tonalidade_gray(datas_vista_fundo, len_fptr_vista_fundo, 9);
	unsigned char * data_gray_toggle_vista_fundo = tonalidade_toggle_gray(datas_vista_fundo, len_fptr_vista_fundo, 3840, 9);
	unsigned char * data_inverte_vista_fundo = inverte_image(datas_vista_fundo, len_fptr_vista_fundo, 9);

	unsigned char * data_gray_vista_lateral_direita = tonalidade_gray(datas_vista_lateral_direita, len_fptr_vista_lateral_direita, 9);
	unsigned char * data_gray_toggle_vista_lateral_direita = tonalidade_toggle_gray(datas_vista_lateral_direita, len_fptr_vista_lateral_direita, 3840, 9);
	unsigned char * data_inverte_vista_lateral_direita = inverte_image(datas_vista_lateral_direita, len_fptr_vista_lateral_direita, 9);

	unsigned char * data_gray_vista_lateral_esquerda = tonalidade_gray(datas_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda, 9);
	unsigned char * data_gray_toggle_vista_lateral_esquerda = tonalidade_toggle_gray(datas_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda, 3840, 9);
	unsigned char * data_inverte_vista_lateral_esquerda = inverte_image(datas_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda, 9);

	unsigned char * data_gray_vista_superior = tonalidade_gray(datas_vista_superior, len_fptr_vista_superior, 9);
	unsigned char * data_gray_toggle_vista_superior = tonalidade_toggle_gray(datas_vista_superior, len_fptr_vista_superior, 3840, 9);
	unsigned char * data_inverte_vista_superior = inverte_image(datas_vista_superior, len_fptr_vista_superior, 9);

	/*Gravação*/
	grava_arquivo(path_memorial_out_gray, data_gray, len_fptr);
	grava_arquivo(path_memorial_out_toggle, data_gray_toggle, len_fptr);
	grava_arquivo(path_memorial_out_inverte, data_inverte, len_fptr);

	grava_arquivo(path_vista_diagonal_out_gray, data_gray_vista_diagonal, len_fptr_vista_diagonal);
	grava_arquivo(path_vista_diagonal_out_toggle, data_gray_toggle_vista_diagonal, len_fptr_vista_diagonal);
	grava_arquivo(path_vista_diagonal_out_inverte, data_inverte_vista_diagonal, len_fptr_vista_diagonal);

	grava_arquivo(path_vista_frontal_out_gray, data_gray_vista_frontal, len_fptr_vista_frontal);
	grava_arquivo(path_vista_frontal_out_toggle, data_gray_toggle_vista_frontal, len_fptr_vista_frontal);
	grava_arquivo(path_vista_frontal_out_inverte, data_inverte_vista_frontal, len_fptr_vista_frontal);

	grava_arquivo(path_vista_fundo_out_gray, data_gray_vista_fundo, len_fptr_vista_fundo);
	grava_arquivo(path_vista_fundo_out_toggle, data_gray_toggle_vista_fundo, len_fptr_vista_fundo);
	grava_arquivo(path_vista_fundo_out_inverte, data_inverte_vista_fundo, len_fptr_vista_fundo);
	
	grava_arquivo(path_vista_lateral_direita_out_gray, data_gray_vista_lateral_direita, len_fptr_vista_lateral_direita);
	grava_arquivo(path_vista_lateral_direita_out_toggle, data_gray_toggle_vista_lateral_direita, len_fptr_vista_lateral_direita);
	grava_arquivo(path_vista_lateral_direita_out_inverte, data_inverte_vista_lateral_direita, len_fptr_vista_lateral_direita);

	grava_arquivo(path_vista_lateral_esquerda_out_gray, data_gray_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda);
	grava_arquivo(path_vista_lateral_esquerda_out_toggle, data_gray_toggle_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda);
	grava_arquivo(path_vista_lateral_esquerda_out_inverte, data_inverte_vista_lateral_esquerda, len_fptr_vista_lateral_esquerda);
	
	grava_arquivo(path_vista_superior_out_gray, data_gray_vista_superior, len_fptr_vista_superior);
	grava_arquivo(path_vista_superior_out_toggle, data_gray_toggle_vista_superior, len_fptr_vista_superior);
	grava_arquivo(path_vista_superior_out_inverte, data_inverte_vista_superior, len_fptr_vista_superior);

	printf("\n[Tempo Total de execução: %fs]\n", (float) (clock() - tempo)  / CLOCKS_PER_SEC);

	return 0;
}
