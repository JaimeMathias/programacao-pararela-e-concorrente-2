#include "hip/hip_runtime.h"
#include "imagem_process.h"
#define RGB 3

#ifdef INSTALL_CUDA

unsigned char * cuda_malloc_managed_aux(unsigned int len)
{
	unsigned char * data;
	if ((data = (unsigned char *)malloc(sizeof(unsigned char *))) == NULL) {
		printf("Erro de Alocação de Memória 0x01\n");
		exit(1);
	}
	hipMallocManaged(&data, sizeof(unsigned char) * len);

	if (data == NULL) {
		printf("Erro na alocação da memória 0x01\n");
		exit(1);
	}
}

__global__
void cuda_tonalidade_gray(unsigned char * data, unsigned char * data_aux, unsigned int header_size, unsigned int n)
{	
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 3 + header_size;
	if (i < n) {
		data_aux[i] = (int) ((0.299 * data[i]) + (0.587 * data[i + 1]) + (0.144 * data[i + 2])); //calcula o valor para conversão
		data_aux[i + 1] = data_aux[i]; //copia o valor para
		data_aux[i + 2] = data_aux[i];  //todas componentes

		if (data_aux[i] > 255) {
			data_aux[i] = 255;
			data_aux[i + 1] = 255;
			data_aux[i + 2] = 255;
		}
	} 
}

__global__
void cuda_inverte_image(unsigned char * data, unsigned char * data_aux,  unsigned int header_size, unsigned int n)
{
	int i = (blockIdx.x * blockDim.x + threadIdx.x);
	if (i < n) {
		data_aux[header_size + i] = data[n - i];
	}
}


__global__
void cuda_tonalidade_toogle_gray(unsigned char * data, unsigned char * data_aux, unsigned int header_size, unsigned int n, unsigned int largura)
{	
	int i = (blockIdx.x * blockDim.x + threadIdx.x) * 3 + header_size, k = 0;

	int largura_rgb = largura * 3;
	int divisao_res = (int) (i / largura_rgb);

	k = i;
	// if (divisao_res % 2 == 0) {
	// 	k += n / 2;
	// }

	// if (k < n) {
	if (k < n && divisao_res % 2 != 0) {
		data_aux[k] = (int) ((0.299 * data[k]) + (0.587 * data[k + 1]) + (0.144 * data[k + 2])); //calcula o valor para conversão
		data_aux[k + 1] = data_aux[k]; //copia o valor para
		data_aux[k + 2] = data_aux[k];  //todas componentes

			if (data_aux[k] > 255) {
				data_aux[k] = 255;
				data_aux[k + 1] = 255;
				data_aux[k + 2] = 255;
			}
	}
}

#endif

void print_data(unsigned char * data, unsigned int len)
{
	unsigned int i, k = 0;
	for (i = 0; i < len; i++, k++) {
		printf("%d. %c , %d\n", i, data[i], data[i]);
	}
	exit(1);
}

FILE * open_arquivo(char * str, char * modo) {

    FILE * arq; //Arquivo lógico
    if ((arq = fopen(str, modo)) == NULL) {
        printf("Erro na abertura do arquivo %s\n", "filename");
        exit(0x3);
    }
    return arq;
}

unsigned long get_len_fptr(FILE * f)
{
	fseek(f, 0, SEEK_END);
	unsigned long len = (unsigned long)ftell(f);
	fseek(f, SEEK_SET, 0);
	return len;
}

void read_file_bin(FILE* f, unsigned char * ptr, unsigned long len_esperado)
{
	unsigned long  qtd_bytes_read;
	qtd_bytes_read = fread(ptr, sizeof(unsigned char), len_esperado, f);
	if (qtd_bytes_read != len_esperado) {
		printf("Erro na Leitura do arquivo\n");
		printf("Número de bytes lidos %ld\n", qtd_bytes_read);
	} else
		printf("Leitura realizada com sucesso\n");
}

void grava_arquivo(char * path, unsigned char * ptr, int tamanho)
{
	FILE * f = open_arquivo(path, "wb");
    unsigned long qtd_bytes_gravados;
    qtd_bytes_gravados = fwrite(ptr, sizeof(unsigned char), tamanho, f);
    if(qtd_bytes_gravados != tamanho) { // verifica se a gravacao funcionou
	printf("Erro na gravacao do arquivo!\n");
	exit(1);
    } else
	printf("Gravacao realizada com sucesso! (%ld)\n", qtd_bytes_gravados);
}

unsigned char * read_image_input(char * _arq, unsigned long * _len_fptr)
{
	FILE* arq = open_arquivo(_arq, "rb");
	*_len_fptr = get_len_fptr(arq);
	printf("O tamanho do arquivo %s é %ld bytes.\n", _arq, *_len_fptr);

	unsigned char *ptr;
	//ptr =  (unsigned char *) malloc(sizeof(unsigned char) * *_len_fptr);
	hipMallocManaged(&ptr, sizeof(unsigned char) * *_len_fptr);
	if (ptr == NULL) {
		printf("Erro na alocação da memória\n");
		exit(1);
	}
	read_file_bin(arq, ptr, *_len_fptr);
	fclose(arq);
	return ptr;
}

void config_params_date_aux(unsigned char * data_aux, unsigned char * data, unsigned int len_header)
{
	for (int i = 0; i < len_header; i++)
	{
		data_aux[i] = data[i];
	}
}

unsigned char * tonalidade_gray(unsigned char * data, unsigned int len_img, unsigned int x)
{
	unsigned char * data_aux = cuda_malloc_managed_aux(len_img);
	
	config_params_date_aux(data_aux, data, DEFAULT_HEADER_SIZE + x);
	unsigned int N = DEFAULT_HEADER_SIZE + x;

	printf("Aplicando tonalidade gray!\n");
	
	#ifdef INSTALL_CUDA

		int block_size = 1024;
		int len_img_pixel = (len_img - N) / 3;
		int num_blocks = (len_img_pixel + block_size - 1) / block_size;
		cuda_tonalidade_gray<<<num_blocks, block_size >>>(data, data_aux, N, len_img);
		hipDeviceSynchronize();

	#endif

	#ifdef INSTALL_CPU

		for (unsigned int i = N; i < len_img; i += RGB) {
			data_aux[i] = (int) ((0.299 * data[i]) + (0.587 * data[i + 1]) + (0.144 * data[i + 2])); //calcula o valor para conversão
			data_aux[i + 1] = data_aux[i]; //copia o valor para
			data_aux[i + 2] = data_aux[i];  //todas componentes

			if (data_aux[i] > 255) {
				data_aux[i] = 255;
				data_aux[i + 1] = 255;
				data_aux[i + 2] = 255;
			}
		}
	#endif

	return data_aux;
}

unsigned char * inverte_image(unsigned char * data, unsigned int len_img, unsigned int x)
{
	unsigned char * data_aux = cuda_malloc_managed_aux(len_img);
	
	config_params_date_aux(data_aux, data, DEFAULT_HEADER_SIZE + x);
	unsigned int N = DEFAULT_HEADER_SIZE + x;

	printf("Aplicando inverte_image!\n");
	
	#ifdef INSTALL_CUDA

		int block_size = 1024;
		int len_img_pixel = len_img - N;
		int num_blocks = (len_img_pixel + block_size - 1) / block_size;
		cuda_inverte_image<<<num_blocks, block_size >>>(data, data_aux, N, len_img);
		hipDeviceSynchronize();

	#endif;

	#ifdef INSTALL_CPU
	
	for (unsigned int i = len_img, j = DEFAULT_HEADER_SIZE + x; i > 14; i--, j++)
	{
		data_aux[j] = data[i];
	}

	#endif

	return data_aux;
}

unsigned char * tonalidade_toggle_gray(unsigned char * data, unsigned int len_img, unsigned largura, unsigned int x)
{
	unsigned char * data_aux = cuda_malloc_managed_aux(len_img);

	config_params_date_aux(data_aux, data, len_img);
	unsigned int N = DEFAULT_HEADER_SIZE + x;

	printf("Aplicando tonalidade gray toogle!\n");

	#ifdef INSTALL_CUDA

		int block_size = 1024;
		// int len_img_pixel = ((len_img - N) / 2) / 3;
		int len_img_pixel = (len_img - N) / 3;
		int num_blocks = (len_img_pixel + block_size - 1) / block_size;
		cuda_tonalidade_toogle_gray<<<num_blocks, block_size>>>(data, data_aux, N, len_img, largura);
		hipDeviceSynchronize();

	#endif

	#ifdef INSTALL_CPU
	bool state_linha = true;
	for (unsigned int i = DEFAULT_HEADER_SIZE + x, k = 0; i < len_img; i += RGB) {
		k = i;
		if (state_linha) {
			data_aux[i] = (int) ((0.299 * data[i]) + (0.587 * data[i + 1]) + (0.144 * data[i + 2])); //calcula o valor para conversão
			data_aux[i + 1] = data_aux[i]; //copia o valor para
			data_aux[i + 2] = data_aux[i];  //todas componentes

	        if (data_aux[i] > 255) {
	            data_aux[i] = 255;
	            data_aux[i + 1] = 255;
	            data_aux[i + 2] = 255;
			}

			if (k > largura * RGB) {
				state_linha = false;
				k = 0;
			}
		}
		else {
			if (k > largura * RGB) {
				state_linha = true;
				k = 0;
			}
		}
	}
	#endif

	return data_aux;
}
