#include "hip/hip_runtime.h"
#include "main.h"


typedef struct {
	FILE * fptr;
}path_arq; //struct para captura da matriz de entrada

 ///* Regiao de Variaves Globais no Escopo main.c *
static lst_ptr_th colun_date[QTD_COLLUN];
static char arq_origem[] = "/home/grupo04/Categorizacao-Dataset/dataset/dataset_00_1000_sem_virg.csv";
//static pthread_mutex_t mutex;

static controles control_process;

static char*** dataset_data;
static char*** dataset_normalizado;
static path_arq path_arq_t[1];
args_memory my_set_memory;



static void calloc_memory_dataset(unsigned int n)
{
    dataset_data =  (char ***) calloc(n, sizeof(char **));
    dataset_normalizado = calloc(n, sizeof(char **));
    unsigned int i;

    #ifdef INSTALL_OMP
        #pragma omp parallel for
    #endif // INSTALL_OMP
    for (i = 0; i < n; i++) {
        dataset_data[i] = (char**) calloc(QTD_COLLUN, sizeof(char *));
        dataset_normalizado[i] = (char**) calloc(QTD_COLLUN, sizeof(char *));
    }
    #ifdef INSTALL_OMP
        #pragma omp parallel for
    #endif // INSTALL_OMP
    for (i = 0; i < n; i++) {
        for (unsigned int j = 0; j < QTD_COLLUN; j++) {
            dataset_data[i][j] =  (char*) calloc(QTD_WORD, sizeof(char));
            dataset_data[i][j][0] = 0;
            dataset_normalizado[i][j] =  (char*) calloc(QTD_WORD, sizeof(char));
            dataset_normalizado[i][j][0] = 0;
        }
    }
}

static void clear_memory_dataset(void * _args)
{
    #ifndef INSTALL_OMP
        #pragma cmp parallel for
        unsigned int i;
        for (i = 0; i < N; i++) {
            for (unsigned int j = 0; j < QTD_COLLUN; j++) {
                strcpy(dataset_normalizado[i][j], "");
                strcpy(dataset_data[i][j], "");
            }
        }
    #endif // INSTALL_OMP
    #ifdef INSTALL_OMP
    args * _m = (args *) _args;
    lst_ptr l = _m->lista;
    while (l != NULL) {
        for (unsigned int j = 0; j < QTD_COLLUN; j++) {
            int debug = l->dado;
            strcpy(dataset_data[l->dado][j], "");
            strcpy(dataset_normalizado[l->dado][j], "");
            //lst_print(l);
            //printf("\n");
        }
        l = l->prox;
    }
    #endif // INSTALL_OMP
}

static void * solicitacao_arquivo_job(void * _args)
{
    args * _args_t = (args*) _args;

    lst_ptr l = _args_t->lista;

    while (l != NULL) {
        lst_ptr_th p = colun_date[l->dado - 1];
        while (p != NULL) {
            if (p->dado.id == 0)
                fprintf(_args_t->fptr_destinos[(l->dado - 1) % QTD_COLLUN_THREAD], "%s,,\n", p->dado.word);
            fprintf(_args_t->fptr_destinos[(l->dado - 1) % QTD_COLLUN_THREAD], "%d,%s,%d\n", p->dado.id, p->dado.word, p->dado.count);
            p = p->prox;
        }
        l = l->prox;
    }
    return 0;
}

/// *Adicionar infos das colunas do arquivo de entrada*
static void add_lst_info_distinct(lst_ptr_th * l, char * str)
{
    int id_t = 0;
	lst_info_th info_t;
	strcpy(info_t.word, str);

	if (!lst_existing_th(*l, info_t, &id_t)) {
		info_t.id = id_t;
        info_t.count = 1;
		lst_ins_th(l, info_t);
	}
}

/// *Verifica se info da coluna é um job a processar*
static bool is_my_job(lst_ptr l, int colun)
{
    while (l != NULL) {
        if (l->dado == colun)
            return true;
        l = l->prox;
    }
    return false;
}

/// *normaliza info separadamente do arquivo de entrada principal*
static int normalize_info_date(args args_t, char * str, int colun, int * _id_word)
{
    if (strcmp(str, "") != 0) {
        lst_info_th info_t;
        strcpy(info_t.word, str);

        if (is_my_job(args_t.lista, colun)) {
            int id = lst_info_id_th(colun_date[colun - 1], info_t);
            if (id != NOT_EXIST) {
                *_id_word = id;
                return PROCEED;
            }
            else
                return HOLD;
        }
        return NEXT;
    }
    else return HOLD;
}

/// *normaliza info unico do arquivo de entrada em unica saida principal*
static void * normaliza_colun_date(void * _args)
{
    //pthread_mutex_lock(&mutex);
	args * args_t = (args*) _args;
	char word[200];
	tipoDado i, id_word, count = 0;

    do {
        lst_ptr l = args_t->lista;
        sem_wait(&control_process.mutexs_threads[args_t->id - 1]);
        sem_wait(&control_process.mutexs_process[args_t->id - 1]);
        while (l != NULL) {
        #ifdef INSTALL_OMP
            #pragma omp parallel for
        #endif // INSTALL_OMP
            for (i = 0; i < N;) {
                switch (normalize_info_date(*args_t, dataset_data[i][l->dado - 1], l->dado, &id_word)) {
                    case HOLD :
                    break;
                    case PROCEED :
                        if (id_word == 0) {
                            sprintf(word, "%s,", dataset_data[i][l->dado - 1]);
                            strcpy(dataset_normalizado[i][l->dado - 1], word);
                        }
                        else {
                            sprintf(word, "%d,", id_word);
                            strcpy(dataset_normalizado[i][l->dado - 1], word);
                        }
                        i += 1;
                    break;
                    case NEXT :
                        i += 1;
                    break;
                 }
            }
            l = l->prox;
        }
        count += i;
        sem_post(&control_process.mutexs_process[args_t->id - 1]);
    } while (count < N_TOTAL);
    //pthread_mutex_unlock(&mutex);
    return;
}

/// *Lidar com controle de escrita do arquivo de saida principal*
static void * solicitacao_arquivo(void * argsArq)
{
	ptr_args_arq _argssArq = (ptr_args_arq) argsArq;
	tipoDado i, j, count = 0;

    do {
        sem_wait(&control_process.mutexs_threads[NUM_THREADS]);
        sem_wait(&control_process.mutexs_process[NUM_THREADS]);
        for (i = 0; i < N; i++) {
            for (j = 0; j < QTD_COLLUN;) {
                if (strcmp(dataset_normalizado[i][j], "") != 0) {
                    fprintf(_argssArq->arq_main, "%s", dataset_normalizado[i][j]);
                    j++;
                }
            }
            fprintf(_argssArq->arq_main, "\n");
        }
        count += i;
        sem_post(&control_process.mutexs_process[NUM_THREADS]);
    } while (count < N_TOTAL);
    fclose(_argssArq->arq_main);
	return 0;
}


static void aguarda_processos_threads()
{
    for (unsigned int i = 0; i < NUM_THREADS + 1; i++)
        sem_wait(&control_process.mutexs_process[i]);
}

static void liberar_processos_threads()
{
    for (unsigned int i = 0; i < NUM_THREADS + 1; i++)
        sem_post(&control_process.mutexs_process[i]);
}

static void desbloqueio_threads()
{
    for (unsigned int i = 0; i < NUM_THREADS + 1; i++)
        sem_post(&control_process.mutexs_threads[i]);
}

static void clear_memory()
{
    #ifdef INSTALL_OMP
    int status;
    for (unsigned int i = 0; i < NUM_THREADS; i++) {
        if (status_create( status = pthread_create((&my_set_memory._my_set[i].thread), NULL, clear_memory_dataset, (void *)&my_set_memory._my_set[i])));
        else exit(0xF);
	}
    #endif // INSTALL_OMP
    #ifndef INSTALL_OMP
     clear_memory_dataset(NULL);
    #endif // INSTALL_OMP

}

static void * ler_matriz_entrada(void * args)
{
	path_arq * _path_arq_t = (path_arq*) args;
	char str[1001], *token;
	unsigned int i;
	tipoDado count = 0;

	for (i = 0; i < QTD_COLLUN; i++)
        lst_init_th(&colun_date[i]);
    do {
        if (count != 0 ) {
            //printf("Aguardando Todos\n");
            aguarda_processos_threads();
            //printf("Liberando Todos\n");
            clear_memory();
            liberar_processos_threads();
            desbloqueio_threads();
        }
        for (i = 0; fscanf(_path_arq_t->fptr, " %500[^\n]s", str) != EOF && i < N; i++) {
            token = strtok(str, ",");
            for (int j = 0; token != NULL && j < QTD_COLLUN; j++) {
                strcpy(dataset_data[i][j], token);
                add_lst_info_distinct(&colun_date[j], token);
                token = strtok(NULL, ",");
            }
        }
        count += i;
    } while (count < N_TOTAL - 1);

	return;
}


int main ()
{
	int i, status;
	clock_t tempo;
	args _args[NUM_THREADS]; //numero de args por threads de CPU
	struct args_arq args_main;
	pthread_t thread_1; //thread responsavel pelo arquivo de entrada

    create_threads(_args, NUM_THREADS, arq_origem, &args_main, &control_process);
    thread_jobs(_args, QTD_COLLUN, NUM_THREADS, &args_main); //repassa trabalhos
    create_threads_mmory_set(&my_set_memory, N);
    //print_responsabilidade_thread(my_set_memory._my_set);
    print_responsabilidade_thread(_args);

    path_arq_t[0].fptr = open_arquivo(arq_origem, "r"); //path dataset

    tempo = clock();
    printf("\nEm execucao ...\n");

    calloc_memory_dataset(N);
    if (status_create( status = pthread_create((&thread_1), NULL, ler_matriz_entrada, (void *)&path_arq_t[0])));
    else exit(0xF);

    if (status_create( status = pthread_create((&args_main.thread), NULL, solicitacao_arquivo, (void *)&args_main)));
    else exit(0xF);

    /*Repassa função de trabalho*/
	for(i = 0; i < NUM_THREADS; i++) {
		if (status_create(status = pthread_create((&_args[i].thread), NULL, normaliza_colun_date, (void *)&_args[i])));
		else exit(0xFF);
	}

    /*Thread principal aguarda todas as thredes de trabalhos finalizarem*/
	for(i = 0; i < NUM_THREADS; i++) {
		pthread_join(_args[i].thread, NULL);
	}
	pthread_join(thread_1, NULL);
	pthread_join(args_main.thread, NULL);
	printf("cheguei aqui\n");

	for (i = 0; i < NUM_THREADS; i++) {
        if (status_create( status = pthread_create((&_args[i].thread), NULL, solicitacao_arquivo_job, (void *)&_args[i])));
        else exit(0xF);
	}
    /*Thread principal aguarda todas as thredes de trabalhos finalizarem*/
	for(i = 0; i < NUM_THREADS; i++) {
		pthread_join(_args[i].thread, NULL);
	}

    printf("\nTerminando processo ...\n");
	printf("\n\n[Tempo Total Do Processo: %fs]\n", (float) (clock() - tempo)  / CLOCKS_PER_SEC);
	return 0;
}
