/* lista.c
 *
 * Implementação das operações sobre o TAD lista ordenada implementada
 * de forma encadeada.
 *
 *
 */

#include "lista.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

void lst_init(lst_ptr * l) {
    *l = NULL;
}

void lst_ins(lst_ptr * l, lst_info_int val) {
    lst_ptr n;
    if ((n = (lst_ptr) malloc(sizeof(struct lst_no_int))) == NULL) {
        fprintf(stderr, "Erro de alocacao de memoria!\n");
        exit(1);
    }
    n->dado = val;
    if (*l == NULL) {
        n->prox = *l;
        *l = n;
        return;
    }
    else {
        lst_ptr p = *l;
         while (p->prox != NULL) {
            p = p->prox;
         }
         n->prox = p->prox;
         p->prox = n;
         return;
    }
}

void lst_print(lst_ptr l) {
    printf("[ ");
    while (l != NULL) {
        printf("%d ,", l->dado);
        l = l->prox;
    }
    printf(" ]\n");
}
