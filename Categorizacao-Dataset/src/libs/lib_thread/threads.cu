/* threads.c
 *
 * Definição das operações sobre o TAD threads e implementação das funçoes especificas dessa TAD.
 */

#include <stdlib.h>
#include "threads.h"


void print_responsabilidade_thread(args * _args)
{
	int i;
	printf("\n____________________________________________________\n");
	for(i = 0; i < NUM_THREADS; i++) {
		printf("\nMeu iDThread: %d\n", _args[i].id);
		printf("As colunas que fiquei responsavel foram :)\n");
		lst_print(_args[i].lista);
	}
}

bool status_create(int status)
{
	if (status) {
		printf("Erro criando thread, retornou codigo %d\n", status );
		return false;
	}
	return true;
}


void create_threads_mmory_set(args_memory * _m, unsigned int n)
{
    unsigned int i;
	for(i = 0; i < NUM_THREADS; i++) {
        lst_init(&_m->_my_set[i].lista);
	}

	for (i = 0; i < n; i++) {
        lst_ins(&_m->_my_set[i % NUM_THREADS].lista, i);
	}
}

void create_threads(args * _args, int n, char * arq_origem, ptr_args_arq _args_main, controles * _control_process)
{
    int i;
    /*Repassa o identificador para as threads*/
	for(i = 0; i < n; i++) {
		lst_init(&_args[i].lista);
		_args[i].id = i + 1;
		sem_init(&_control_process->mutexs_threads[i], 0, 1);
		sem_init(&_control_process->mutexs_process[i], 0, 1);
	}
	sem_init(&_control_process->mutexs_threads[i], 0, 1);
    sem_init(&_control_process->mutexs_process[i], 0, 1);
	_args_main->id = i;
}

void thread_jobs(args * _args, int n , int n_threads, ptr_args_arq _args_main)
{
    int i, j, qtd_colun = 0, n_thread = 0;
    char str_path[100];

    /*Repassa linha de trabalho ciclica, por thread*/
	while (qtd_colun < QTD_COLLUN - 1) {
            //_args[n_thread].id = n_thread;
            _args[n_thread].main_destino = _args_main;
        for (j = 0; j < 4; j++) {
            qtd_colun += 1;
            lst_ins(&_args[n_thread].lista, qtd_colun);
            sprintf(str_path, "/home/grupo04/Categorizacao-Dataset/dataset/colun-%d.csv", qtd_colun);
            _args[n_thread].fptr_destinos[j] = open_arquivo(str_path, "w");
        }
        n_thread += 1;
	}
	for (i = 0; i < QTD_COLLUN_THREAD; i++)
        for (j = 0; j < NUM_THREADS; j++) {
            _args_main->thread_buffer[i][j].state = TO_DO;
        }
	_args_main->arq_main = open_arquivo("/home/grupo04/Categorizacao-Dataset/dataset/principal-normalizado.csv", "w");
}

FILE *open_arquivo(char * str, char * modo) {

    FILE * arq; //Arquivo lógico
    if ((arq = fopen(str, modo)) == NULL) {
        printf("Erro na abertura do arquivo %s\n", "filename");
        exit(0x3);
    }
    return arq;
}
