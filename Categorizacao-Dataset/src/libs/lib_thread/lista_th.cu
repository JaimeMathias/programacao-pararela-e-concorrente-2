
/* lista_th.c
 *
 */

#include "lista_th.h"


void lst_init_th(lst_ptr_th * l) {
    *l = NULL;
}

void lst_ins_th(lst_ptr_th * l, lst_info_th val) {

    lst_ptr_th n;
    if ((n = (lst_ptr_th) malloc(sizeof(struct lst_no_th))) == NULL) {
        fprintf(stderr, "Erro de alocacao de memoria!\n");
        exit(1);
    }
    n->dado = val;
    if (*l == NULL) {
        n->prox = *l;
        *l = n;
        return;
    }
    else {
        lst_ptr_th p = *l;
         while (p->prox != NULL) {
            p = p->prox;
         }
         n->prox = p->prox;
         p->prox = n;
         return;
    }
}

void lst_print_th(lst_ptr_th l) {
    printf("[ ");
    while (l != NULL) {
        printf("[%s,%d] ", l->dado.word, l->dado.id);
        l = l->prox;
    }
    printf(" ]\n");
}

bool lst_existing_th(lst_ptr_th l, lst_info_th x, int * id)
{
    int count = -1;
    while (l != NULL) {
        if (!strcmp(l->dado.word, x.word)) {
                l->dado.count++;
                return true;
        }
        l = l->prox;
        count += 1;
    }
    *id = count + 1;
    return false;
}

unsigned int lst_info_id_th(lst_ptr_th l, lst_info_th x)
{
    while (l != NULL) {
        if (!strcmp(l->dado.word, x.word))
            return l->dado.id;
        l = l->prox;
    }
    return NOT_EXIST;
}
